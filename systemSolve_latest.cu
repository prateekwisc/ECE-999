#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2009-2012 EM Photonics, Inc.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to EM Photonics ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code may
 * not redistribute this code without the express written consent of EM
 * Photonics, Inc.
 *
 * EM PHOTONICS MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR IMPLIED
 * WARRANTY OF ANY KIND.  EM PHOTONICS DISCLAIMS ALL WARRANTIES WITH REGARD TO
 * THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF MERCHANTABILITY,
 * NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.  IN NO EVENT SHALL EM
 * PHOTONICS BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, OR CONSEQUENTIAL
 * DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS OF USE, DATA OR
 * PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE OR OTHER TORTIOUS
 * ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE OR PERFORMANCE OF THIS
 * SOURCE CODE.  
 *
 * U.S. Government End Users.   This source code is a "commercial item" as that
 * term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of "commercial
 * computer  software"  and "commercial computer software documentation" as
 * such terms are  used in 48 C.F.R. 12.212 (SEPT 1995) and is provided to the
 * U.S. Government only as a commercial end item.  Consistent with 48
 * C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 227.7202-4 (JUNE 1995), all
 * U.S. Government End Users acquire the source code with only those rights set
 * forth herein. 
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code, the
 * above Disclaimer and U.S. Government End Users Notice.
 *
 */

/*
 * CULA Example: systemSolve
 *
 * This example shows how to use a system solve for multiple data types.  Each
 * data type has its own example case for clarity.  For each data type, the
 * following steps are done:
 *
 * 1. Allocate a matrix on the host
 * 2. Initialize CULA
 * 3. Initialize the A matrix to the Identity
 * 4. Call gesv on the matrix
 * 5. Verify the results
 * 6. Call culaShutdown
 *
 * After each CULA operation, the status of CULA is checked.  On failure, an
 * error message is printed and the program exits.
 *
 * Note: CULA Premium and double-precision GPU hardware are required to run the
 * double-precision examples
 *
 * Note: this example performs a system solve on an identity matrix against a
 * random vector, the result of which is that same random vector.  This is not
 * true in the general case and is only appropriate for this example.  For a
 * general case check, the product A*X should be checked against B.  Note that
 * because A is modifed by GESV, a copy of A would be needed with which to do
 * the verification.
 */


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include<hip/hip_runtime.h>
#include <cula_lapack.h>
#include<cula_lapack_device.h>

void checkStatus(culaStatus status)
{
    char buf[256];

    if(!status)
        return;

    culaGetErrorInfoString(status, culaGetErrorInfo(), buf, sizeof(buf));
    printf("%s\n", buf);

    culaShutdown();
    exit(EXIT_FAILURE);
}


void culaDeviceFloatExample()
{
#ifdef NDEBUG
    int N = 20;
#else
    int N = 1024;
#endif
    int NRHS = 1;
    int i, j;
hipEvent_t start, stop;
float t_sgesv;

hipEventCreate(&start);

hipEventCreate(&stop);

    culaStatus status;
    
    culaFloat* A = NULL;
    culaFloat* A_bak = NULL;
    culaFloat* B = NULL;
    culaFloat* X = NULL;
    culaInt* IPIV = NULL;


    culaDeviceFloat* Ad = NULL;
    culaDeviceFloat* Ad_bak = NULL;
    culaDeviceFloat* Bd = NULL;
    culaDeviceFloat* Xd = NULL;
    culaDeviceInt* IPIVd = NULL;

     
     
    // culaFloat one = 2.0f;
    culaFloat thresh = 1e-6f;
    culaFloat diff;

    printf("-------------------\n");
    printf("       SGESV\n");
    printf("-------------------\n");

    printf("Allocating Matrices on host\n");
    A = (culaFloat*)malloc(N*N*sizeof(culaFloat));
    A_bak = (culaFloat*)malloc(N*N*sizeof(culaFloat));
    B = (culaFloat*)malloc(N*sizeof(culaFloat));
    X = (culaFloat*)malloc(N*sizeof(culaFloat));
    IPIV = (culaInt*)malloc(N*sizeof(culaInt));
    if(!A || !B || !IPIV || !A_bak)
        exit(EXIT_FAILURE);

    printf("Allocating Matrices on device\n");
    hipMalloc((void**)&Ad,N*N*sizeof(culaFloat));
    // hipMalloc((void**)&Ad_bak,N*N*sizeof(culaFloat));
    hipMalloc((void**)&Bd,N*sizeof(culaFloat));
    hipMalloc((void**)&Xd,N*sizeof(culaFloat));
    hipMalloc((void**)&IPIVd,N*sizeof(culaInt));

hipEventRecord(start, 0);

hipDeviceProp_t deviceProp;
int devcount;
hipGetDeviceCount(&devcount);
printf("Available Cards: ");
for(int i = 0; i< devcount; ++i)
{
//hipDeviceProp_t deviceProp;
//const int currentDevice = 1;
if(hipGetDeviceProperties(&deviceProp, i) == hipSuccess)
  printf("Device %d: %s\n", i, deviceProp.name);
}

const int currentDevice = 1;

if(hipGetDeviceProperties(&deviceProp, currentDevice) == hipSuccess)
printf("CULA is currently using Device %d: %s\n", currentDevice, deviceProp.name);

    printf("Initializing CULA\n");
    status = culaInitialize();
    checkStatus(status);

    // Set A to the identity matrix
    memset(A, 0, N*N*sizeof(culaFloat));
    memset(A_bak, 0, N*N*sizeof(culaFloat));
    for(i = 0; i < N; ++i) {
      A_bak[i*N+i] = A[i * N + i] = 1.f;
     // printf("%g, %g\n", one, A[i * N + i]);
    }
  /* A[0]=3;
   A[1]=3;
   A[2]=0;
   A[3]=0;
   A[4]=2;
   A[5]=2;
   A[6]=1;
   A[7]=0;
   A[8]=1;*/
    //Printing the matix
    
    // Set B to a random matrix (see note at top)
    for(i = 0; i < N; ++i)
        B[i] = (culaFloat)(rand());
    memcpy(X, B, N*sizeof(culaFloat));

    memset(IPIV, 0, N*sizeof(culaInt));

//Copy from Host to Device
    hipMemcpy(Ad,A, N*N*sizeof(culaFloat),hipMemcpyHostToDevice);
    // hipMemcpy(Ad_bak,A_bak, N*N*sizeof(culaFloat),hipMemcpyHostToDevice);
    hipMemcpy(Bd,B, N*sizeof(culaFloat),hipMemcpyHostToDevice);
    hipMemcpy(Xd,X, N*sizeof(culaFloat),hipMemcpyHostToDevice);
    hipMemcpy(IPIVd,IPIV, N*sizeof(culaInt),hipMemcpyHostToDevice);


    //Printing the matix
    printf("\n The Matrix A is: \n");
      for (i=0; i< N*N; ++i)
      {
      printf("%g,",A[i]);
      if((i+1)%N==0)
        printf("\n");
      }
    printf("Calling culaSgesv\n");
    status = culaDeviceSgesv(N, NRHS, Ad, N, IPIVd, Xd, N);
    checkStatus(status);
//Copy result from Device to Host

    hipMemcpy(A,Ad, N*N*sizeof(culaFloat),hipMemcpyDeviceToHost);
    // hipMemcpy(A_bak,Ad_bak, N*N*sizeof(culaFloat),hipMemcpyDeviceToHost);
    hipMemcpy(B,Bd, N*sizeof(culaFloat),hipMemcpyDeviceToHost);
    hipMemcpy(X,Xd, N*sizeof(culaFloat),hipMemcpyDeviceToHost);
    hipMemcpy(IPIV,IPIVd, N*sizeof(culaInt),hipMemcpyDeviceToHost);




    printf("Verifying Result\n");
    int success = 1;
    float max_b = 0.0;
for(i =0; i< N; i++){

  diff = X[i] - B[i];
//  if(max_b < fabs(B[i]))
 //   max_b = fabs(B[i]);
  if(diff < max_b)
    diff = -diff;
  if(diff > thresh)
    printf("Result check failed:  i=%d  X[i]=%f  B[i]=%f\n", i, X[i],B[i]);
}

    for(i = 0; i < N; ++i)
    {
      fprintf(stderr, "X[%d] = %g, B[%d] = %g\n", i, X[i], i, B[i]);
    }
    
    if (success)
      printf("Success\n");
    else
      printf("Failed\n");
    
    printf("Shutting down CULA\n\n");
    culaShutdown();
hipEventRecord(stop, 0);
hipEventSynchronize(stop);

//printf("\n Time taken for CULA Sgesv is %f", t_sgesv);
    free(A);
    free(A_bak);
    free(X);
    free(B);
    free(IPIV);
    hipFree(Ad);
    hipFree(Ad_bak);
    hipFree(Bd);
    hipFree(Xd);
    hipFree(IPIVd);

hipEventElapsedTime(&t_sgesv, start, stop);

printf("\n Time taken for CULA Sgesv is %f ms\n", t_sgesv);
}


void culaDeviceFloatComplexExample()
{
#ifdef NDEBUG
    int N = 4096;
#else
    int N = 512;
#endif
    int NRHS = 1;
    int i;

    culaStatus status;
    
    culaFloatComplex* A = NULL;
    culaFloatComplex* B = NULL;
    culaFloatComplex* X = NULL;
    culaInt* IPIV = NULL;

    culaFloatComplex one = { 1.0f, 0.0f };
    culaFloat thresh = 1e-6f;
    culaFloat diffr;
    culaFloat diffc;
    culaFloat diffabs;

    printf("-------------------\n");
    printf("       CGESV\n");
    printf("-------------------\n");

    printf("Allocating Matrices\n");
    A = (culaFloatComplex*)malloc(N*N*sizeof(culaFloatComplex));
    B = (culaFloatComplex*)malloc(N*sizeof(culaFloatComplex));
    X = (culaFloatComplex*)malloc(N*sizeof(culaFloatComplex));
    IPIV = (culaInt*)malloc(N*sizeof(culaInt));
    if(!A || !B || !IPIV)
        exit(EXIT_FAILURE);

    printf("Initializing CULA\n");
    status = culaInitialize();
    checkStatus(status);

    // Set A to the identity matrix
    memset(A, 0, N*N*sizeof(culaFloatComplex));
    for(i = 0; i < N; ++i)
        A[i*N+i] = one;
    
    // Set B to a random matrix (see note at top)
    for(i = 0; i < N; ++i)
    {
        B[i].x = (culaFloat)rand();
        B[i].y = (culaFloat)rand();
    }
    memcpy(X, B, N*sizeof(culaFloatComplex));

    memset(IPIV, 0, N*sizeof(culaInt));

    printf("Calling culaCgesv\n");
    status = culaCgesv(N, NRHS, A, N, IPIV, X, N);
    checkStatus(status);

    printf("Verifying Result\n");
    for(i = 0; i < N; ++i)
    {
        diffr = X[i].x - B[i].x;
        diffc = X[i].y - B[i].y;
        diffabs = (culaFloat)sqrt(X[i].x*X[i].x+X[i].y*X[i].y)
                - (culaFloat)sqrt(B[i].x*B[i].x+B[i].y*B[i].y);
        if(diffr < 0.0f)
            diffr = -diffr;
        if(diffc < 0.0f)
            diffc = -diffc;
        if(diffabs < 0.0f)
            diffabs = -diffabs;
        if(diffr > thresh || diffc > thresh || diffabs > thresh)
            printf("Result check failed:  i=%d  X[i]=(%f,%f)  B[i]=(%f,%f)", i, X[i].x, X[i].y, B[i].x, B[i].y);
    }
    
    printf("Shutting down CULA\n\n");
    culaShutdown();


    free(A);
    free(B);
    free(IPIV);
}


// Note: CULA Premium is required for double-precision
#ifdef CULA_PREMIUM
void culaDeviceDoubleExample()
{
#ifdef NDEBUG
    int N = 800;
#else
    int N = 512;
#endif
    int NRHS = 1;
    int i,j;
hipEvent_t start, stop;
float t_dgesv;

hipEventCreate(&start);

hipEventCreate(&stop);

    culaStatus status;
    
    culaDouble* A = NULL;
    culaDouble* A_bak = NULL;
    culaDouble* B = NULL;
    culaDouble* X = NULL;
    culaInt* IPIV = NULL;

    culaDeviceDouble* Ad = NULL;
    culaDeviceDouble* Ad_bak = NULL;
    culaDeviceDouble* Bd = NULL;
    culaDeviceDouble* Xd = NULL;
    culaDeviceInt* IPIVd = NULL;



//    culaDouble *work = NULL;
    // culaDouble *swork = NULL;
//    int *info;

//    culaDouble one = 1.0;
    culaDouble thresh = 1e-6;
    culaDouble diff;
    
    printf("\t-------------------\n");
    printf("       DGESV\n");
    printf("-------------------\n");

    printf("Allocating Matrices\n");
    A = (culaDouble*)malloc(N*N*sizeof(culaDouble)); 
    A_bak = (culaDouble*)malloc(N*N*sizeof(culaDouble));
    B = (culaDouble*)malloc(N*sizeof(culaDouble));
    X = (culaDouble*)malloc(N*sizeof(culaDouble));
    IPIV = (culaInt*)malloc(N*sizeof(culaInt));
  //  work = (culaDouble*)malloc(N * NRHS * sizeof(culaDouble));
    //swork = (culaDouble*)malloc(N * (N+NRHS) * sizeof(culaDouble));
//    info = (int *)malloc(N * sizeof(int));
    if(!A || !B || !IPIV || !A_bak)
        exit(EXIT_FAILURE);


    printf("Allocating Matrices on device\n");
    hipMalloc((void**)&Ad,N*N*sizeof(culaDouble));
  //  hipMalloc((void**)&Ad_bak,N*N*sizeof(culaFloat));
    hipMalloc((void**)&Bd,N*sizeof(culaDouble));
    hipMalloc((void**)&Xd,N*sizeof(culaDouble));
    hipMalloc((void**)&IPIVd,N*sizeof(culaInt));

hipEventRecord(start, 0);
    
    printf("Initializing CULA\n");
    status = culaInitialize();
    checkStatus(status);

    // Set A to the identity matrix
    memset(A, 0, N*N*sizeof(culaDouble));
    memset(A_bak, 0, N*N*sizeof(culaDouble));
    for(i = 0; i < N; ++i){
      A_bak[i * N + i] = A[i*N + i] = 2.f;  
    if(i > 0)
      A_bak[i * N + i-1] = A[i*N + i-1] = 0.5f;
    if(i < N-1)
      A_bak[i * N + i+1] = A[i*N + i + 1] = 0.5f;
    }
    // Set B to a random matrix (see note at top
    for(i = 0; i < N; ++i)
        B[i] = (culaDouble)(rand() % 10);
    memcpy(X, B, N*sizeof(culaDouble));

    memset(IPIV, 0, N*sizeof(culaInt));

//Copy from Host to Device
    hipMemcpy(Ad,A, N*N*sizeof(culaDouble),hipMemcpyHostToDevice);
//    hipMemcpy(Ad_bak,A_bak, N*N*sizeof(culaFloat),hipMemcpyHostToDevice);
    hipMemcpy(Bd,B, N*sizeof(culaDouble),hipMemcpyHostToDevice);
    hipMemcpy(Xd,X, N*sizeof(culaDouble),hipMemcpyHostToDevice);
    hipMemcpy(IPIVd,IPIV, N*sizeof(culaInt),hipMemcpyHostToDevice);

    printf("Calling culaDgesv\n");
    int iter = 0;
    status = culaDeviceDgesv(N, NRHS, Ad, N, IPIVd, Xd, N);
   // printf("iter = %d\n", iter);
    if(status == culaInsufficientComputeCapability)
    {
        printf("No Double precision support available, skipping example\n");
        free(A);
        free(B);
        free(IPIV);
        culaShutdown();
        return;
    }
    checkStatus(status);

    
//Copy result from Device to Host

    hipMemcpy(A,Ad, N*N*sizeof(culaDouble),hipMemcpyDeviceToHost);
//    hipMemcpy(A_bak,Ad_bak, N*N*sizeof(culaFloat),hipMemcpyDeviceToHost);
    hipMemcpy(B,Bd, N*sizeof(culaDouble),hipMemcpyDeviceToHost);
    hipMemcpy(Xd,X, N*sizeof(culaDouble),hipMemcpyDeviceToHost);
    hipMemcpy(IPIVd,IPIV, N*sizeof(culaInt),hipMemcpyDeviceToHost);

    printf("Verifying Result\n");
    int success = 1;
    double max_b = 0.0;
    for (i = 0; i < N; i++)
      if (max_b < fabs(B[i]))
        max_b = fabs(B[i]);

/*    for(i = 0; i < N; ++i)
    {
      fprintf(stderr, "X[%d] = %g,B[%d] = %g\n", i, X[i], i, B[i]);
    }*/
    if(success)
      printf("Success\n");
    else
      printf("Failed\n");

    printf("Shutting down CULA\n\n");
    culaShutdown();
hipEventRecord(stop, 0);
hipEventSynchronize(stop);

    free(A);
    free(A_bak);
    free(X);
    free(B);
    free(IPIV);
    hipFree(Ad);
    hipFree(Ad_bak);
    hipFree(Bd);
    hipFree(Xd);
    hipFree(IPIVd);
hipEventElapsedTime(&t_dgesv, start, stop);

printf("\n Time taken for CULA Dgesv is %f ms \n", t_dgesv);

}


void culaDoubleComplexExample()
{
#ifdef NDEBUG
    int N = 1024;
#else
    int N = 128;
#endif
    int NRHS = 1;
    int i;

    culaStatus status;
    
    culaDoubleComplex* A = NULL;
    culaDoubleComplex* B = NULL;
    culaDoubleComplex* X = NULL;
    culaInt* IPIV = NULL;

    culaDoubleComplex one = { 1.0, 0.0 };
    culaDouble thresh = 1e-6;
    culaDouble diffr;
    culaDouble diffc;
    culaDouble diffabs;

    printf("-------------------\n");
    printf("       ZGESV\n");
    printf("-------------------\n");

    printf("Allocating Matrices\n");
    A = (culaDoubleComplex*)malloc(N*N*sizeof(culaDoubleComplex));
    B = (culaDoubleComplex*)malloc(N*sizeof(culaDoubleComplex));
    X = (culaDoubleComplex*)malloc(N*sizeof(culaDoubleComplex));
    IPIV = (culaInt*)malloc(N*sizeof(culaInt));
    if(!A || !B || !IPIV)
        exit(EXIT_FAILURE);

    printf("Initializing CULA\n");
    status = culaInitialize();
    checkStatus(status);

    // Set A to the identity matrix
    memset(A, 0, N*N*sizeof(culaDoubleComplex));
    for(i = 0; i < N; ++i)
        A[i*N+i] = one;
    
    // Set B to a random matrix (see note at top)
    for(i = 0; i < N; ++i)
    {
        B[i].x = (culaDouble)rand();
        B[i].y = (culaDouble)rand();
    }
    memcpy(X, B, N*sizeof(culaDoubleComplex));

    memset(IPIV, 0, N*sizeof(culaInt));

    printf("Calling culaZgesv\n");
    status = culaZgesv(N, NRHS, A, N, IPIV, X, N);
    if(status == culaInsufficientComputeCapability)
    {
        printf("No Double precision support available, skipping example\n");
        free(A);
        free(B);
        free(IPIV);
        culaShutdown();
        return;
    }
    checkStatus(status);

    printf("Verifying Result\n");
    for(i = 0; i < N; ++i)
    {
        diffr = X[i].x - B[i].x;
        diffc = X[i].y - B[i].y;
        diffabs = (culaDouble)sqrt(X[i].x*X[i].x+X[i].y*X[i].y)
                - (culaDouble)sqrt(B[i].x*B[i].x+B[i].y*B[i].y);
        if(diffr < 0.0)
            diffr = -diffr;
        if(diffc < 0.0)
            diffc = -diffc;
        if(diffabs < 0.0)
            diffabs = -diffabs;
        if(diffr > thresh || diffc > thresh || diffabs > thresh)
            printf("Result check failed:  i=%d  X[i]=(%f,%f)  B[i]=(%f,%f)", i, X[i].x, X[i].y, B[i].x, B[i].y);
    }
    
    printf("Shutting down CULA\n\n");
    culaShutdown();

    free(A);
    free(B);
    free(IPIV);
}
#endif


int main(int argc, char** argv)
{
    culaDeviceFloatExample();
    // culaFloatComplexExample();
    
    // Note: CULA Premium is required for double-precision
#ifdef CULA_PREMIUM
//    culaDeviceDoubleExample();
  //  culaDoubleComplexExample();
#endif

    return EXIT_SUCCESS;
}

